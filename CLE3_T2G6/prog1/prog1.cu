#include "hip/hip_runtime.h"

/**
 *  \file prog1.cu (implementation file)
 *
 *  \brief Problem name: Bitonic Sort Row Processing.
 *
 *
 *  \authors Manuel Diaz & Tiago Santos - June 2024
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include <hip/hip_runtime.h>

/**
 *   program configuration
 */

# define N 1024 * 1024

# define K 1024


/* returns the number of seconds elapsed between the two specified times */
static double get_delta_time(void);

/* returns 1 if the specified array is sorted, and 0 otherwise */
int validateSort(int *arr, int n);

/* Function to merge two haves of array */
__device__ void merge(int arr[], int l, int m, int r);

/* Iterative mergesort */
__device__ void merge_sort(int arr[], int n);

/* kernel function */
__global__ void process_sequence_sorting(int *data, int iter);


/**
 *  \brief Function merge.
 *
 *  This function merges two sorted subarrays into a single sorted subarray.
 *
 *  \param array: pointer to the array containing the subarrays
 *  \param left: starting index of the first subarray
 *  \param mid: ending index of the first subarray and starting index of the second subarray
 *  \param right: ending index of the second subarray
 *
 *  The function creates temporary arrays to store the subarrays and then merges them into the original 
 *  array in a sorted order.
 */
__device__ void merge(int array[], int left, int mid, int right)
{
    int i, j, k;
    int n1 = mid - left + 1;
    int n2 =  right - mid;

	int *L = (int*)malloc(n1 * sizeof(int));
	int *R = (int*)malloc(n2 * sizeof(int));
 
    // Copy data to temporary arrays
    for (i = 0; i < n1; i++)
        L[i] = array[left + i];
    for (j = 0; j < n2; j++)
        R[j] = array[mid + 1+ j];
 
    // Merge temporary arrays into arr
    i = 0;
    j = 0;
    k = left;
    while (i < n1 && j < n2) {
        if (L[i] <= R[j]) {
            array[k] = L[i];
            i++;
        } else {
            array[k] = R[j];
            j++;
        }
        k++;
    }
 
    // Copy remaining elements of L[]
    while (i < n1) {
        array[k] = L[i];
        i++;
        k++;
    }
 
    // Copy remaining elements of R[]
    while (j < n2) {
        array[k] = R[j];
        j++;
        k++;
    }

	free(L);
	free(R);
}

/**
 *  \brief Function merge_sort.
 *
 *  This function sorts an array using the merge sort algorithm.
 *
 *  \param array: pointer to the array to be sorted
 *  \param size: size of the array
 *
 *  The function divides the array into smaller subarrays and recursively sorts them using merge sort. 
 *  It then merges the sorted subarrays to obtain the final sorted array.
 */
__device__ void merge_sort(int array[], int size) {
   int currentSize, leftStart;
	
	for (currentSize = 1; currentSize <= size - 1; currentSize = 2 * currentSize) {
		for (leftStart = 0; leftStart < size - 1; leftStart += 2 * currentSize) {
           int middle = min(leftStart + currentSize - 1, size - 1);
           int rightEnd = min(leftStart + 2 * currentSize - 1, size - 1);
           merge(array, leftStart, middle, rightEnd);
       	}
   	}
}

/**
 *  \brief Function process_sequence_sorting.
 *
 *  This CUDA kernel function performs parallel processing on the input array using merge sort algorithm.
 *
 *  \param data: pointer to the input array
 *  \param iter: iteration number indicating the level of merge sort
 *
 *  The function divides the input array into subsequences and sorts them using merge sort.
 *  Each thread is responsible for sorting a specific subsequence.
 *  In each iteration, the function performs either an independent merge sort on a subsequence (when iter is 0) 
 *  or merges two previously sorted subsequences.
 */
__global__ void process_sequence_sorting(int *data, int iter) {
	//int N = DIM;
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = blockDim.x * gridDim.x * y + x;

    int limit = K >> iter;
	if(idx >= limit)
        return;
    
	int start = N/K * (1 << iter) * idx;
	int end = start + (1 << iter) * N/K;
	int mid = (start + end) / 2;
	int subseq_len = (1 << iter) * N/K;
	int *subseq_start = data + start;

	(iter == 0) ? merge_sort(subseq_start, subseq_len) : merge(data, start, mid-1, end-1);
    __syncthreads();
}

/**
 *  \brief Validate Sort.
 *
 *  This function checks if an array is sorted in ascending order.
 *
 *  \param arr: pointer to the array to be validated
 *  \param n: size of the array
 *
 */
int validateSort(int *arr, int n) {
    int i;

    for (i = 0; i < n - 1; i++)
    {
        if (arr[i] > arr[i + 1])
        {
            printf("Error in position %d between element %d and %d\n", i, arr[i], arr[i + 1]);
            return 0;
        }
    }
	if (i == (n - 1))
		printf("Everything is OK!\n");

    return 1;
}


/**
 *  \brief Main function.
 *
 *  \param argc: number of command-line arguments
 *  \param argv: array of command-line argument strings
 *
 *  The function reads an input file containing integers, performs parallel merge sort using CUDA, 
 *  and validates the sorted array.
 */
int main (int argc, char **argv)
{
	if (argc != 2) {
		printf("Usage: %s <filename>\n", argv[0]);
		return 1;
	}

	/* Open the file for reading */

	FILE *file = fopen(argv[1], "rb");
	if (file == NULL) {
		printf("Failed to open file: %s\n", argv[1]);
		return 1;
	}

	fseek(file, 0, SEEK_END);
	int size = ftell(file) / sizeof(int);
	fseek(file, 0, SEEK_SET);

	int *host_matrix = (int*) malloc(size * sizeof(int));
	if (host_matrix == NULL) {
		printf("Error: cannot allocate memory\n");
		return 1;
	}

	int count = fread(host_matrix, sizeof(int), size, file);

	if (count != size) {
		printf("Error: could not read all integers from file\n");
		return 1;
	}

	fclose(file);


	/* set up the device */

	int dev = 0;

	hipDeviceProp_t deviceProp;
	CHECK (hipGetDeviceProperties (&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK (hipSetDevice (dev));

	/* copy the host data to the device memory */
	int *device_matrix;
	CHECK(hipMalloc((void**)&device_matrix, N * sizeof(int)));
	CHECK(hipMemcpy(device_matrix, host_matrix, K * sizeof(int[K]), hipMemcpyHostToDevice));


	/* launch the kernel */

	int gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ;

	// Number of threads in each dimension of a block
	blockDimX = 1 << 0;                                             // optimize!
	blockDimY = 1 << 0;                                             // optimize!
	blockDimZ = 1 << 0;                                             // do not change!

	// Number of blocks in each dimension of the grid
	gridDimX = K;													// optimize!
	gridDimY = 1 << 0;												// optimize!
	gridDimZ = 1 << 0;                                              // do not change!

	dim3 grid (gridDimX, gridDimY, gridDimZ);
	dim3 block (blockDimX, blockDimY, blockDimZ);

	if ((gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ) != K) {
		printf ("Wrong configuration!\n");
		printf("blockDimX = %d, blockDimY = %d, blockDimZ = %d\n", blockDimX, blockDimY, blockDimZ);
		printf("gridDimX = %d, gridDimY = %d, gridDimZ = %d\n", gridDimX, gridDimY, gridDimZ);
		return 1;
	}

	// Perform merge sort
	(void) get_delta_time ();

    int iter = 0; 
    int size2 = (N / K) * (1 << iter);

	if (K == 1) {
		printf("Iteration = %d\n", iter);

		process_sequence_sorting<<<grid, block>>>(device_matrix, iter);
		gridDimX = K / (1 << (iter + 1));
	}
	else {
		for (int iter = 0; size2 < N; iter++) {
        	printf("Iteration = %d\n", iter);

			process_sequence_sorting<<<grid, block>>>(device_matrix, iter);
			gridDimX = K / (1 << (iter + 1));  // Divides by 2 each iteration

			dim3 grid (gridDimX, gridDimY, gridDimZ);
        	size2 = (N / K) * (1 << iter);

			CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
			CHECK (hipGetLastError ());                                 // check for kernel errors
		}
	}
	
	CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
	CHECK (hipGetLastError ());                                 // check for kernel errors

	printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3f seconds to run\n",
			gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time ());

	/* copy kernel result back to host side */
	CHECK (hipMemcpy (host_matrix, device_matrix, K * sizeof(int[K]), hipMemcpyDeviceToHost));

	/* free device global memory */
	CHECK (hipFree (device_matrix));

	/* reset the device */
	CHECK (hipDeviceReset ());

	// validate if the array is sorted correctly
	validateSort(host_matrix, K);
	free(host_matrix);
	return 0;
}

/**
 *  \brief Get delta time.
 *
 *  This function measures the elapsed time between successive calls.
 *
 *  \return The time elapsed between successive calls in seconds.
 *
 *  The function uses the CLOCK_MONOTONIC clock to measure time.
 */
static double get_delta_time(void)
{
	static struct timespec t0,t1;

	t0 = t1;
	if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
	{
		perror("clock_gettime");
		exit(1);
	}
	return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}